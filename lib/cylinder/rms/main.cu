#include "hip/hip_runtime.h"
//
//  main.cpp
//  diffusion_cylinder_exchange
//
//  Update Journal:
//  -- 6/26/2017: massive job version
//  -- 7/20/2017: do not divide DW signal & cumulants by b0 signal, record particle number in each compartment
//  -- 4/27/2019: diffusion in spheres with permeable membrane
//  -- 5/1/2019: implement cuda
//  -- 6/18/2019: re-write sphere code to cylinder code for cuda
//  -- 2/20/2020: re-write coaxial cylinder code to single-layer cylinder code for cuda
//  -- 3/6/2020: fix the bug for permeation step along z-axis
//  -- 4/23/2020: replace pow with pow2, turn on translateFlag
//
//  Created by Hong-Hsi Lee in February, 2017.
//


#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <math.h>
#include <iomanip>
#include <time.h>
#include <cstdlib>
#include <algorithm>
#include <string>
#include <complex>
#include <string>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

using namespace std;
    
#define Pi 3.14159265
#define timepoints 1000
#define nbin 200
#define nite 100

// ********** diffusion library **********

__device__ double pow2 (const double &x) {
    return (x*x);
}

__device__ void pixPosition ( const double x_in[], const unsigned int &NPix, int xPix[] ) {
    double x[2]={0}; x[0]=x_in[0]; x[1]=x_in[1]; //x[2]=x_in[2];
    
    if ( x[0]<0 ) { x[0]+=1; }
    if ( x[0]>1 ) { x[0]-=1; }
    
    if ( x[1]<0 ) { x[1]+=1; }
    if ( x[1]>1 ) { x[1]-=1; }
    
//    if ( x[2]<0 ) { x[2]+=1; }
//    if ( x[2]>1 ) { x[2]-=1; }
    
    xPix[0]=floor(x[0]*NPix);
    xPix[1]=floor(x[1]*NPix);
//    xPix[2]=floor(x[2]*NPix);
}

__device__ void translateXc ( const double x[], double xc[] ) {
    // Translate circle center xc to make it as close to the position x as possible
    double ti=0, tj=0;
    double d2 = pow2(x[0]-xc[0])+pow2(x[1]-xc[1]), d2Tmp=0;
    int ii[2]={0}, jj[2]={0};
    ii[1]=2*(xc[0]<0.5)-1;
    jj[1]=2*(xc[1]<0.5)-1;
    for (int i=0; i<2; i++) {
        for (int j=0; j<2; j++) {
            if ( i==0 & j==0 ){ continue; }
            d2Tmp=pow2(x[0]-xc[0]-ii[i])+pow2(x[1]-xc[1]-jj[j]);
            if (d2Tmp<d2) {
                d2=d2Tmp;
                ti=ii[i];
                tj=jj[j];
            }
        }
    }
    xc[0]+=ti;
    xc[1]+=tj;
}

__device__ bool inCyl ( const double x[], const double xc_in[], const double &rc, const bool &translateFlag ) {
    double xc[2]={0}; xc[0]=xc_in[0]; xc[1]=xc_in[1];
    // If the point x is in the circle (xc,rc), return 1; if not, return 0.
    
    // Translate circle center xc to make it as close to the position xt as possible
    if ( translateFlag ) { translateXc(x,xc); }
    
    return ( ( pow2(x[0]-xc[0])+pow2(x[1]-xc[1]) ) <= rc*rc );
}

__device__ bool stepE2A (const double xi[], const double xt[], const double xc_in[], const double &rc, double t[], const double &dx, const bool &translateFlag) {
    double xc[2]={0}; xc[0]=xc_in[0]; xc[1]=xc_in[1];
    // If segment(xi,xt) overlaps circle (xc,rc), return 1; if not, return 0.
    
    // Translate circle center xc to make it as close to the position xt as possible
    if ( translateFlag ) { translateXc(xt,xc); }
    
    t[0]=-( (xi[0]-xc[0])*(xt[0]-xi[0])+(xi[1]-xc[1])*(xt[1]-xi[1]) )/dx/dx;
    
    // If xt is in the cell, segment overlaps the circle.
    if ( ( pow2(xt[0]-xc[0])+pow2(xt[1]-xc[1]) ) <= rc*rc ) {
        return true;
    } else {
        // L: a line connecting xi and xt
        // xl: a point on L closest to xc, xl = xi + (xt-xi)*t
        // d: distance of xc to L (or xl)
        // Reference: http://mathworld.wolfram.com/Point-LineDistance3-Dimensional.html
        double xl[2]={0};
        xl[0]=xi[0]+(xt[0]-xi[0])*t[0];
        xl[1]=xi[1]+(xt[1]-xi[1])*t[0];
        double d2=pow2(xl[0]-xc[0])+pow2(xl[1]-xc[1]);
        
        // If d>rc, segment does not overlap the circle.
        if (d2>rc*rc) {
            return false;
        } else {
            // xl is in ICS, but xi and xt are both in ECS.
            return ( ( (xi[0]-xl[0])*(xt[0]-xl[0])+(xi[1]-xl[1])*(xt[1]-xl[1]) ) <= 0 );
        }
    }
}

__device__ void elasticECS (const double x[], const double v[], const double &dx, const double &dz, const double xc_in[], const double &rc, const bool &translateFlag, double xt[]) {
    double xc[2]={0}; xc[0]=xc_in[0]; xc[1]=xc_in[1];
    // Elastic collision from x in ECS onto a cell membrane (xc,rc) with a direction v and a step size dx.
    
    // Translate circle center xc to make it as close to the position (x + dx*v) as possible
    double xTmp[3]={0};
    xTmp[0]=x[0]+dx*v[0];
    xTmp[1]=x[1]+dx*v[1];
    xTmp[2]=x[2]+dz*v[2];
    if ( translateFlag ) { translateXc(xTmp,xc); }
    
    // distance( x+t*v, xc )==rc, solve t
    double a=0,b=0,c=0,t1=0,t2=0,t=0;
    a=v[0]*v[0] + v[1]*v[1];
    b=2*(x[0]-xc[0])*v[0] + 2*(x[1]-xc[1])*v[1];
    c=pow2(x[0]-xc[0]) + pow2(x[1]-xc[1]) - rc*rc;
    
    
    // xt: final position, xm: contact point on cell membrane, n: unit normal vector
    // discri: discriminant
    double xm[2]={0}, n[2]={0};
    double discri=b*b-4*a*c;
    if (discri<=0) {                    // Does not encounter the cell membrane
        xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
    } else {
        discri=sqrt(discri);
        t1=0.5/a*( -b+discri );
        t2=0.5/a*( -b-discri );
        t=min(t1,t2);
        if ( (t>=dx) | (t<0) ) {        // Does encounter the cell membrane
            xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
        } else {                          // Encounter the cell membrane
            // xm = x + t*v;
            xm[0]=x[0]+t*v[0];
            xm[1]=x[1]+t*v[1];
            
            // n parallel to (xm-xc), outward unit normal vector
            t1=sqrt( pow2(xc[0]-xm[0])+pow2(xc[1]-xm[1]) );
            n[0]=(xm[0]-xc[0])/t1;
            n[1]=(xm[1]-xc[1])/t1;
            
            // v' = v - 2*dot(v,n)*n
            t1=v[0]*n[0]+v[1]*n[1];
            n[0]=v[0]-2*t1*n[0];
            n[1]=v[1]-2*t1*n[1];
            
            // xt = xm + (dx-t)*v'
            xt[0]=xm[0]+(dx-t)*n[0];
            xt[1]=xm[1]+(dx-t)*n[1];
            xt[2]=xTmp[2];
        }
    }
}

__device__ void permeateE2I (const double x[], const double v[], const double &dxEX, const double &dzEX, const double xc_in[], const double &rc, const double &dxIN, const double &dzIN, const bool &translateFlag, double xt[]) {
    double xc[2]={0}; xc[0]=xc_in[0]; xc[1]=xc_in[1];
    // Permeation from x in ECS into a cell (xc,rc) with a direction v and a step size dxEX.
    
    // Translate circle center xc to make it as close to the position (x + dx*v) as possible
    double xTmp[3]={0};
    xTmp[0]=x[0]+dxEX*v[0];
    xTmp[1]=x[1]+dxEX*v[1];
    xTmp[2]=x[2]+dzEX*v[2];
    if ( translateFlag ) { translateXc(xTmp,xc); }
    
    // distance( x+t*v, xc )==rc, solve t
    double a=0,b=0,c=0,t1=0,t2=0,t=0;
    a=v[0]*v[0] + v[1]*v[1];
    b=2*(x[0]-xc[0])*v[0] + 2*(x[1]-xc[1])*v[1];
    c=pow2(x[0]-xc[0]) + pow2(x[1]-xc[1]) - rc*rc;
    
    // xt: final position, xm: contact point on cell membrane, n: unit normal vector
    // discri: discriminant
    double xm[2]={0}, n[2]={0};
    double discri=b*b-4*a*c;
    if (discri<=0) {                    // Does not encounter the cell membrane
        xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
    } else {
        discri=sqrt(discri);
        t1=0.5/a*( -b+discri );
        t2=0.5/a*( -b-discri );
        t=min(t1,t2);
        if ( (t>=dxEX) | (t<0) ) {      // Does encounter the cell membrane
            xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
        } else {                          // Encounter the cell membrane
            // xm = x + t*v;
            xm[0]=x[0]+t*v[0];
            xm[1]=x[1]+t*v[1];
            
            // n parallel to (xc-xm), inward unit normal vector
            t1=sqrt( pow2(xc[0]-xm[0])+pow2(xc[1]-xm[1]) );
            n[0]=(xc[0]-xm[0])/t1;
            n[1]=(xc[1]-xm[1])/t1;
            
            // Diffuse transverse to the cell membrane after permeation to make the result converge faster.
            // xt = xm + n*(1-t/dxEX)*dxIN*dot(v,n)
            t1=fabs(v[0]*n[0]+v[1]*n[1]);
            xt[0]=xm[0]+n[0]*(1-t/dxEX)*dxIN*t1;
            xt[1]=xm[1]+n[1]*(1-t/dxEX)*dxIN*t1;
            // xt = t/dxEX*dzEX*vz + (1-t/dxEX)*dzIN*vz
            xt[2]=x[2] + t/dxEX*dzEX*v[2] + (1-t/dxEX)*dzIN*v[2];
        }
    }
}

__device__ void elasticICS (const double x[], const double v[], const double &dx, const double &dz, const double xc_in[], const double &rc, const bool &translateFlag, double xt[]) {
    double xc[2]={0}; xc[0]=xc_in[0]; xc[1]=xc_in[1];
    // Elastic collision from x in ICS onto a cell membrane (xc,rc) with a direction v and a step size dx.
    
    // Translate circle center xc to make it as close to the position (x + dx*v) as possible
    double xTmp[3]={0};
    xTmp[0]=x[0]+dx*v[0];
    xTmp[1]=x[1]+dx*v[1];
    xTmp[2]=x[2]+dz*v[2];
    if ( translateFlag ) { translateXc(xTmp,xc); }
    
    // distance( x+t*v, xc )==rc, solve t
    double a=0,b=0,c=0,t1=0,t2=0,t=0;
    a=v[0]*v[0] + v[1]*v[1];
    b=2*(x[0]-xc[0])*v[0] + 2*(x[1]-xc[1])*v[1];
    c=pow2(x[0]-xc[0]) + pow2(x[1]-xc[1]) - rc*rc;

    // xt: final position, xm: contact point on cell membrane, n: unit normal vector
    // discri: discriminant
    double xm[2]={0}, n[2]={0};
    double discri=b*b-4*a*c;
    if (discri<=0) {                     // Walker is right on the surface and diffuses tangent to the surface
//        xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
        xt[0]=x[0]; xt[1]=x[1]; xt[2]=xTmp[2];
    } else {
        discri=sqrt(discri);
        t1=0.5/a*( -b+discri );
        t2=0.5/a*( -b-discri );
        t=max(t1,t2);
        if ( t>=dx ) {                  // Does not encounter the cell membrane
            xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
        }
        else {                          // Encounter the cell membrane
            // xm = x + t*v;
            xm[0]=x[0]+t*v[0];
            xm[1]=x[1]+t*v[1];
            
            // n parallel to (xm-xc), outward unit normal vector
            t1=sqrt( pow2(xc[0]-xm[0])+pow2(xc[1]-xm[1]) );
            n[0]=(xm[0]-xc[0])/t1;
            n[1]=(xm[1]-xc[1])/t1;
            
            // v' = v - 2*dot(v,n)*n
            t1=v[0]*n[0]+v[1]*n[1];
            n[0]=v[0]-2*t1*n[0];
            n[1]=v[1]-2*t1*n[1];
            
            // xt = xm + (dx-t)*v'
            xt[0]=xm[0]+(dx-t)*n[0];
            xt[1]=xm[1]+(dx-t)*n[1];
            xt[2]=xTmp[2];
        }
    }
}

__device__ void permeateI2E (const double x[], const double v[], const double &dxIN, const double &dzIN, const double xc_in[], const double &rc, const double &dxEX, const double &dzEX, const bool &translateFlag, double xt[]) {
    double xc[2]={0}; xc[0]=xc_in[0]; xc[1]=xc_in[1];
    // Permeation from x in ICS out of the cell (xc,rc) with a direction v and a step size dxIN.
    
    // Translate circle center xc to make it as close to the position (x + dx*v) as possible
    double xTmp[3]={0};
    xTmp[0]=x[0]+dxIN*v[0];
    xTmp[1]=x[1]+dxIN*v[1];
    xTmp[2]=x[2]+dzIN*v[2];
    if ( translateFlag ) { translateXc(xTmp,xc); }
    
    // distance( x+t*v, xc )==rc, solve t
    double a=0,b=0,c=0,t1=0,t2=0,t=0;
    a=v[0]*v[0] + v[1]*v[1];
    b=2*(x[0]-xc[0])*v[0] + 2*(x[1]-xc[1])*v[1];
    c=pow2(x[0]-xc[0]) + pow2(x[1]-xc[1]) - rc*rc;
    
    // xt: final position, xm: contact point on cell membrane, n: unit normal vector
    // discri: discriminant
    double xm[2]={0}, n[2]={0};
    double discri=b*b-4*a*c;
    if (discri<=0) {                     // Walker is right on the surface and diffuses tangent to the surface
        xt[0]=x[0]; xt[1]=x[1]; xt[2]=x[2];
    } else {
        discri=sqrt(discri);
        t1=0.5/a*( -b+discri );
        t2=0.5/a*( -b-discri );
        t=max(t1,t2);
        if ( t>=dxIN ) {                 // Does not encounter the cell membrane
            xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
        } else {                         // Encounter the cell membrane
            // xm = x + t*v;
            xm[0]=x[0]+t*v[0];
            xm[1]=x[1]+t*v[1];
            
            // n parallel to (xm-xc), outward unit normal vector
            t1=sqrt( pow2(xc[0]-xm[0])+pow2(xc[1]-xm[1]) );
            n[0]=(xm[0]-xc[0])/t1;
            n[1]=(xm[1]-xc[1])/t1;
            
            // Diffuse perpendicular to the cell membrane after permeation to make the result converge faster.
            // xt = xm + n*(1-t/dxIN)*dxEX*dot(v,n)
            t1=fabs(v[0]*n[0]+v[1]*n[1]);
            xt[0]=xm[0]+n[0]*(1-t/dxIN)*dxEX*t1;
            xt[1]=xm[1]+n[1]*(1-t/dxIN)*dxEX*t1;
            // xt = t/dxIN*dzIN*vz + (1-t/dxIN)*dzEX*vz
            xt[2]=x[2] + t/dxIN*dzIN*v[2] + (1-t/dxIN)*dzEX*v[2];
        }
    }
}

// ********** cuda kernel **********
__device__ double atomAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));
        
        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    
    return __longlong_as_double(old);
}

__global__ void setup_kernel(hiprandStatePhilox4_32_10_t *state, unsigned long seed){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void propagate(hiprandStatePhilox4_32_10_t *state, double *dx2, double *dx4, double *NParICS, double *NParBin, double *sig, const int TN, const int NPar, const int Nbvec, const double res, const double stepIN, const double stepEX, const double stepINz, const double stepEXz, const double probI, const double probE, const unsigned int NPix, const unsigned int Nmax, const int initFlag, const double *xCir, const double *yCir, const double *rCir, const bool *translateFlag, const unsigned int *APix, const double *btab, const double *TD){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandStatePhilox4_32_10_t localstate=state[idx];
    
    int Tstep=TN/timepoints;
    
    for (int k=idx; k<NPar; k+=stride){
        double xPar[3]={0}, xCirTmp[3]={0};
        int xParG[2]={0};                                   // Particle position on a grid
        
        bool instruction1=false, instruction2=false;
        unsigned int a=0, aTmp=0;                           // Element of APix matrix
        int a1=0, a2=0;                                     // Label of cells close to the particle
        int acell[4]={0}; bool instruction[4]={0};          // Cell label
        double tcell[4]={0};                                // xi+(xt-xi)*tcell is a point on segment(xi,xt) closest to the cell center
        double tcellTmp[1]={0};
        double tcellMin=0;
        
        double xi[3]={0}, xt[3]={0}, xTmp[3]={0};           // Particle position
        double xCollision[3]={0};                           // Position after collision
        int xtG[2]={0}, xTmpG[2]={0};                       // Position on grid after diffusion
        double vrand=0;                                     // Random number
        int tidx=0, bidx=0;
        
        double vp[3]={0};                                   // Nomalized diffusion velocity
        int acell_hit=0;                                    // Label of the cell encountered by the walker
        bool iterateFlag=false;                             // true: choose another direction and leap again, false: finish the iteration
        int ite=0;                                          // number of iterations
        bool ICSFlag=false;                                 // true: in ICS, false: not in ICS
        
        int xjmp=0, yjmp=0, zjmp=0;
        double dx=0, dy=0, dz=0;
        
        double qx=0;
        
        //********** Initialize Walker Positions *********
        while (1){
            xPar[0]=hiprand_uniform_double(&localstate);
            xPar[1]=hiprand_uniform_double(&localstate);
            xPar[2]=hiprand_uniform_double(&localstate);
            
            if ( initFlag==1 ) { // 1. Initial positon: ICS
                // Identify the cells close to the walker
                pixPosition(xPar,NPix,xParG);
                a=APix[ NPix*xParG[0]+xParG[1] ];
                a1=a%Nmax; a2=a/Nmax;
                
                // If the walker is in ICS, take the initial position
                instruction1=false; instruction2=false;
                if ( a1 ){
                    xCirTmp[0]=xCir[a1-1]; xCirTmp[1]=yCir[a1-1];
                    instruction1=inCyl(xPar,xCirTmp,rCir[a1-1],translateFlag[a1-1]);
                }
                
                if ( a2 ){
                    xCirTmp[0]=xCir[a2-1]; xCirTmp[1]=yCir[a2-1];
                    instruction2=inCyl(xPar,xCirTmp,rCir[a2-1],translateFlag[a2-1]);
                }
                
                if ( instruction1 || instruction2 ){
                    xi[0]=xPar[0]; xi[1]=xPar[1]; xi[2]=xPar[2];
                    break;
                }
            } else if ( initFlag==2 ) { // 2. Initial positon: ECS
                // Identify the cells close to the walker
                pixPosition(xPar,NPix,xParG);
                a=APix[ NPix*xParG[0]+xParG[1] ];
                a1=a%Nmax; a2=a/Nmax;
                
                // If the walker is not in Axon (ICS+myelin), take the initial position
                instruction1=false; instruction2=false;
                if ( a1 ){
                    xCirTmp[0]=xCir[a1-1]; xCirTmp[1]=yCir[a1-1];
                    instruction1=inCyl(xPar,xCirTmp,rCir[a1-1],translateFlag[a1-1]);
                }
                
                if ( a2 ){
                    xCirTmp[0]=xCir[a2-1]; xCirTmp[1]=yCir[a2-1];
                    instruction2=inCyl(xPar,xCirTmp,rCir[a2-1],translateFlag[a2-1]);
                }
                
                if ( instruction1==false && instruction2==false ){
                    xi[0]=xPar[0]; xi[1]=xPar[1]; xi[2]=xPar[2];
                    break;
                }
            } else if ( initFlag==3 ) { // 3. Initial position: ICS+ECS
                xi[0]=xPar[0]; xi[1]=xPar[1]; xi[2]=xPar[2];
                break;
            } else if ( initFlag==4 ) { // 4. Initial position: center
                xi[0]=0.5; xi[1]=0.5; xi[2]=0.5;
                break;
            }
        }
        
        // ********** Simulate diffusion **********
        xt[0]=xi[0]; xt[1]=xi[1]; xt[2]=xi[2];
        pixPosition(xt,NPix,xtG);                             // Position on grid
        for (int i=0; i<TN; i++){
            // The cells close to the walker in the previous step
            a=APix[ NPix*xtG[0]+xtG[1] ];
            a1=a%Nmax, a2=a/Nmax;
            
            // Check if the particle is in axon
            instruction1=false; instruction2=false;
            if ( a1 ) {
                xCirTmp[0]=xCir[a1-1]; xCirTmp[1]=yCir[a1-1];
                instruction1=inCyl(xt,xCirTmp,rCir[a1-1],translateFlag[a1-1]);
            }
            
            if ( a2 ) {
                xCirTmp[0]=xCir[a2-1]; xCirTmp[1]=yCir[a2-1];
                instruction2=inCyl(xt,xCirTmp,rCir[a2-1],translateFlag[a2-1]);
            }
            
            ICSFlag=false;
            if (instruction1 | instruction2) { ICSFlag=true; }
            
            iterateFlag=true; ite=0;
            
            // ********** One step **********
            while (iterateFlag & (ite<nite)) {
                if ( (instruction1==false) & (instruction2==false) ) {
                    // Case 1 Diffusion In ECS
                    
                    acell[0]=a1; acell[1]=a2;
                    
                    // Primitive position after diffusion
                    vrand=hiprand_uniform_double(&localstate);
                    vp[0]=cos(2*Pi*vrand);
                    vp[1]=sin(2*Pi*vrand);
                    vrand=hiprand_uniform_double(&localstate);
                    vp[2]=2.0*static_cast<double>(vrand<0.5)-1.0;
                    xTmp[0]=xt[0]+stepEX*vp[0];
                    xTmp[1]=xt[1]+stepEX*vp[1];
                    xTmp[2]=xt[2]+stepEXz*vp[2];
                    
                    pixPosition(xTmp,NPix,xTmpG);
                    aTmp=APix[ NPix*xTmpG[0]+xTmpG[1] ];
                    acell[2]=aTmp%Nmax; acell[3]=aTmp/Nmax;
                    
                    // Check if the segment(xt,xTmp) overlaps with any cell
                    for (int j=0; j<4; j++) {
                        instruction[j]=false; tcell[j]=-1;
                        if ( acell[j] ) {
                            xCirTmp[0]=xCir[acell[j]-1];
                            xCirTmp[1]=yCir[acell[j]-1];
                            instruction[j]=stepE2A(xt, xTmp, xCirTmp, rCir[acell[j]-1], tcellTmp, stepEX, translateFlag[acell[j]-1]);
                            tcell[j]=tcellTmp[0];
                        }
                    }
                    if ( (instruction[0]==false) & (instruction[1]==false) & (instruction[2]==false) & (instruction[3]==false) ) {
                        // Case 1.1 Walker diffuses in ECS and does not encounter any cell membrane.
                        xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
                        iterateFlag=false; ICSFlag=false;
                    } else {
                        // Case 1.2 Walker diffuses in ECS and encounters the cell membrane.
                        
                        // Determine the cell to collide with.
                        tcellMin=-1;
                        for (int j=0; j<4; j++) {
                            if ( instruction[j] & (tcell[j]>=0) ) {
                                tcellMin=tcell[j];
                            }
                        }
                        if ( tcellMin<0 ){
                            printf("error: Walker in ECS does not encounter the cell membrane.\n");
                            printf("%d %d %d %d\n",acell[0],acell[1],acell[2],acell[3]);
                            printf("%s %s %s %s\n",instruction[0] ? "true":"false",instruction[1] ? "true":"false",instruction[2] ? "true":"false",instruction[3] ? "true":"false");
                            printf("%.4f %.4f %.4f %.4f\n",tcell[0],tcell[1],tcell[2],tcell[3]);
                        }
                        
                        for (int j=0; j<4; j++) {
                            if ( instruction[j] & (tcell[j]>=0) ) {
                                if (tcell[j]<=tcellMin){
                                    tcellMin=tcell[j];
                                    acell_hit=acell[j];
                                }
                            }
                        }
                        
                        xCirTmp[0]=xCir[acell_hit-1];
                        xCirTmp[1]=yCir[acell_hit-1];
                        
                        vrand=hiprand_uniform_double(&localstate);
                        if (vrand<probE) {
                            // Case 1.2.1 Permeation from ECS to ICS
                            permeateE2I(xt, vp, stepEX, stepEXz, xCirTmp, rCir[acell_hit-1], stepIN, stepINz, translateFlag[acell_hit-1], xTmp);
                            xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
                            iterateFlag=false; ICSFlag=true;
                        }
                        else {
                            // Case 1.2.2 Elastic collision in ECS
                            elasticECS(xt, vp, stepEX, stepEXz, xCirTmp, rCir[acell_hit-1], translateFlag[acell_hit-1], xCollision);
                            
                            // Use xTmp to save the present position
                            xTmp[0]=xt[0]; xTmp[1]=xt[1]; xTmp[2]=xt[2];
                            
                            // Case 1.2.2.1 Renew the step for the elastic collision
                            xt[0]=xCollision[0]; xt[1]=xCollision[1]; xt[2]=xCollision[2];
                            iterateFlag=false;
                            
                            // Case 1.2.2.2 Cancel this step and choose another direction if bouncing twice
                            pixPosition(xCollision, NPix, xTmpG);
                            aTmp=APix[ NPix*xTmpG[0]+xTmpG[1] ];
                            acell[2]=aTmp%Nmax, acell[3]=aTmp/Nmax;
                            
                            if ( acell[2] ) {
                                xCirTmp[0]=xCir[acell[2]-1];
                                xCirTmp[1]=yCir[acell[2]-1];
                                if ( inCyl(xCollision, xCirTmp, rCir[acell[2]-1], translateFlag[acell[2]-1]) ) {
                                    xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
                                    iterateFlag=true; ite++;
                                }
                            }
                            
                            if ( acell[3] ) {
                                xCirTmp[0]=xCir[acell[3]-1];
                                xCirTmp[1]=yCir[acell[3]-1];
                                if ( inCyl(xCollision, xCirTmp, rCir[acell[3]-1], translateFlag[acell[3]-1]) ) {
                                    xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
                                    iterateFlag=true; ite++;
                                }
                            }
                            
                            if ( iterateFlag==false ) {
                                ICSFlag=false;
                            }
                        }
                    }
                } else {
                    // Case 2 Diffusion in ICS
                    if ( instruction1 ){
                        acell[0]=a1;
                    } else if ( instruction2 ){
                        acell[0]=a2;
                    } else {
                        acell[0]=0;
                        printf("error: Walker in ICS has no cell label.\n");
                    }
                    
                    // Primitive position after diffusion
                    vrand=hiprand_uniform_double(&localstate);
                    vp[0]=cos(2*Pi*vrand);
                    vp[1]=sin(2*Pi*vrand);
                    vrand=hiprand_uniform_double(&localstate);
                    vp[2]=2.0*static_cast<double>(vrand<0.5)-1;
                    xTmp[0]=xt[0]+stepIN*vp[0];
                    xTmp[1]=xt[1]+stepIN*vp[1];
                    xTmp[2]=xt[2]+stepINz*vp[2];
                    
//                     pixPosition(xTmp,NPix,xTmpG);
//                     aTmp=APix[ NPix*xTmpG[0]+xTmpG[1] ];
//                     acell[2]=aTmp%Nmax; acell[3]=aTmp/Nmax;
                    
                    // Check if the segment(xt,xTmp) overlaps with the cell membrane
                    xCirTmp[0]=xCir[acell[0]-1];
                    xCirTmp[1]=yCir[acell[0]-1];
                    instruction[0]=inCyl(xTmp,xCirTmp,rCir[acell[0]-1],translateFlag[acell[0]-1]);
                    
                    if ( instruction[0] ) {
                        // Case 2.1 Walker diffuses in ICS and does not encounter the cell membrane
                        xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
                        iterateFlag=false; ICSFlag=true;
                    } else {
                        // Case 2.2 Walker diffuses in ICS and encounters the cell membrane
                        acell_hit=acell[0];
                        xCirTmp[0]=xCir[acell_hit-1];
                        xCirTmp[1]=yCir[acell_hit-1];
                        
                        vrand=hiprand_uniform_double(&localstate);
                        if (vrand<probI) {
                            // Case 2.2.1 Permeation from ICS to ECS
                            permeateI2E(xt, vp, stepIN, stepINz, xCirTmp, rCir[acell_hit-1], stepEX, stepEXz, translateFlag[acell_hit-1], xCollision);
                            
                            // Use xTmp to save the present position
                            xTmp[0]=xt[0]; xTmp[1]=xt[1]; xTmp[2]=xt[2];
                            
                            // Case 2.2.1.1 Renew the step for the permeation
                            xt[0]=xCollision[0]; xt[1]=xCollision[1]; xt[2]=xCollision[2];
                            iterateFlag=false;
                            
                            // Case 2.2.1.2 Cancel this step and choose another direction if steping into another axon
                            pixPosition(xCollision, NPix, xTmpG);
                            aTmp=APix[ NPix*xTmpG[0]+xTmpG[1] ];
                            acell[2]=aTmp%Nmax; acell[3]=aTmp/Nmax;

                            if ( acell[2] ) {
                                xCirTmp[0]=xCir[acell[2]-1];
                                xCirTmp[1]=yCir[acell[2]-1];
                                if ( inCyl(xCollision, xCirTmp, rCir[acell[2]-1], translateFlag[acell[2]-1]) ) {
                                    xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
                                    iterateFlag=true; ite++;
                                }
                            }
                            
                            if ( acell[3] ) {
                                xCirTmp[0]=xCir[acell[3]-1];
                                xCirTmp[1]=yCir[acell[3]-1];
                                if ( inCyl(xCollision, xCirTmp, rCir[acell[3]-1], translateFlag[acell[3]-1]) ) {
                                    xt[0]=xTmp[0]; xt[1]=xTmp[1]; xt[2]=xTmp[2];
                                    iterateFlag=true; ite++;
                                }
                            }
                            
                            if ( iterateFlag==false ){
                                ICSFlag=false;
                            }
                        }
                        else {
                            // Case 2.2.2 Elastic collision in ICS
                            elasticICS(xt, vp, stepIN, stepINz, xCirTmp, rCir[acell_hit-1], translateFlag[acell_hit-1], xCollision);
                            
                            if ( inCyl(xCollision, xCirTmp, rCir[acell_hit-1], translateFlag[acell_hit-1]) ) {
                                // Case 2.2.2.1 Renew the step for the elastic collision
                                xt[0]=xCollision[0]; xt[1]=xCollision[1]; xt[2]=xCollision[2];
                                iterateFlag=false; ICSFlag=true;
                            } else {
                                // Case 2.2.2.2 Cancel this step and choose another direction if bouncing twice
                                iterateFlag=true; ite++;
                            }
                        }
                    }
                }
            }
            pixPosition(xt, NPix, xtG);                // Position on grid after diffusion

            if (ite==nite) {
                printf("Run out of iterations.\n");
            }
            
            // Periodic boundary condition
            if (xt[0]>1) {
                xt[0]-=1;
                xjmp+=1;
            }
            else if (xt[0]<0) {
                xt[0]+=1;
                xjmp-=1;
            }
            
            if (xt[1]>1) {
                xt[1]-=1;
                yjmp+=1;
            }
            else if (xt[1]<0) {
                xt[1]+=1;
                yjmp-=1;
            }
            
            if (xt[2]>1) {
                xt[2]-=1;
                zjmp+=1;
            }
            else if (xt[2]<0) {
                xt[2]+=1;
                zjmp-=1;
            }
            
            // ********** End one step **********
            
            if ( (i%Tstep) ==0 ) { // Save moment tensor for dx^2 and dx^4, and signal for the b-table
                tidx=i/Tstep;
                
                if ( ICSFlag ) { atomAdd(&NParICS[tidx],1); }
                if ( initFlag==4 ) {
                    bidx=floor( sqrt( pow2(xt[0]-0.5) + pow2(xt[1]-0.5) )*nbin*2 );
                    if (bidx<nbin) {
                        atomAdd(&NParBin[nbin*tidx+bidx],1);
                    }
                }
                
                
                dx=(xt[0]+xjmp-xi[0])*res;
                dy=(xt[1]+yjmp-xi[1])*res;
                dz=(xt[2]+zjmp-xi[2])*res;
                
                atomAdd(&dx2[6*tidx+0],dx*dx);
                atomAdd(&dx2[6*tidx+1],dx*dy);
                atomAdd(&dx2[6*tidx+2],dx*dz);
                atomAdd(&dx2[6*tidx+3],dy*dy);
                atomAdd(&dx2[6*tidx+4],dy*dz);
                atomAdd(&dx2[6*tidx+5],dz*dz);
                
                atomAdd(&dx4[15*tidx+0],dx*dx*dx*dx);
                atomAdd(&dx4[15*tidx+1],dx*dx*dx*dy);
                atomAdd(&dx4[15*tidx+2],dx*dx*dx*dz);
                atomAdd(&dx4[15*tidx+3],dx*dx*dy*dy);
                atomAdd(&dx4[15*tidx+4],dx*dx*dy*dz);
                atomAdd(&dx4[15*tidx+5],dx*dx*dz*dz);
                atomAdd(&dx4[15*tidx+6],dx*dy*dy*dy);
                atomAdd(&dx4[15*tidx+7],dx*dy*dy*dz);
                atomAdd(&dx4[15*tidx+8],dx*dy*dz*dz);
                atomAdd(&dx4[15*tidx+9],dx*dz*dz*dz);
                atomAdd(&dx4[15*tidx+10],dy*dy*dy*dy);
                atomAdd(&dx4[15*tidx+11],dy*dy*dy*dz);
                atomAdd(&dx4[15*tidx+12],dy*dy*dz*dz);
                atomAdd(&dx4[15*tidx+13],dy*dz*dz*dz);
                atomAdd(&dx4[15*tidx+14],dz*dz*dz*dz);
                
                for (int j=0; j<Nbvec; j++) {
                    qx=sqrt(btab[4*j]/TD[tidx])*( btab[4*j+1]*dx + btab[4*j+2]*dy + btab[4*j+3]*dz );
                    atomAdd(&sig[Nbvec*tidx+j],cos(qx));
                }
            }
            
        }
    }
    state[idx]=localstate;
}

    
//********** Define tissue parameters **********

int main(int argc, char *argv[]) {
    
    clock_t begin=clock();
    clock_t end=clock();
    
    // Define index number
    int i=0, j=0;
    
    //********** Load mictostructure **********
    
    double dt=0;                // Time step in ms
    int TN=0;                   // Number of time steps
    int NPar=0;                 // Number of time points to record
    int Nbvec=0;                // Number of gradient directions
    
    double Din=0;               // Diffusion coefficient inside the axon in �m^2/ms
    double Dex=0;               // Diffusion coefficient outside the axon in �m^2/ms
    double kappa=0;             // Permeability of a lipid bi-layer in �m/ms
    int initFlag=1;             // Initial position: 1=ICS, 2=ECS, 3=ICS+ECS+myelin, 4=center
    int thread_per_block=0;     // Number of threads per block
    
    unsigned int NPix=0, NAx=0;
    double res=0;
    
    // simulation parameter
    ifstream myfile0 ("simParamInput.txt", ios::in);
    myfile0>>dt; myfile0>>TN; myfile0>>NPar; myfile0>>Nbvec;
    myfile0>>Din; myfile0>>Dex; myfile0>>kappa;
    myfile0>>initFlag;
    myfile0>>thread_per_block;
    myfile0.close();
    
    double stepIN=sqrt(4.0*dt*Din);     // Step size in ICS in �m
    double stepEX=sqrt(4.0*dt*Dex);     // Step size in ECS in �m
    
    double stepINz=sqrt(2.0*dt*Din);
    double stepEXz=sqrt(2.0*dt*Dex);
    
    // resolution
    ifstream myfile1 ("phantom_res.txt", ios::in);
    myfile1>>res;
    myfile1.close();
    
    // Pixel # along each side
    ifstream myfile2 ("phantom_NPix.txt", ios::in);
    myfile2>>NPix;
    myfile2.close();
    
    // Pixelized matrix A indicating axon labels
    thrust::host_vector<unsigned int> APix(NPix*NPix);
    ifstream myfile3 ("phantom_APix.txt", ios::in);
    for (i=0; i<NPix*NPix; i++){
        myfile3>>APix[i];
    }
    myfile3.close();
    
    // Number of axons
    ifstream myfile4 ("phantom_NAx.txt", ios::in);
    myfile4>>NAx;
    myfile4.close();
    
    // Circle center of x-coordinate
    thrust::host_vector<double> xCir(NAx);
    ifstream myfile5 ("phantom_xCir.txt", ios::in);
    for (i=0; i<NAx; i++){
        myfile5>>xCir[i];
    }
    myfile5.close();
    
    // Circle center of y-coordinate
    thrust::host_vector<double> yCir(NAx);
    ifstream myfile6 ("phantom_yCir.txt", ios::in);
    for (i=0; i<NAx; i++){
        myfile6>>yCir[i];
    }
    myfile6.close();
    
    // Circle outer radius
    thrust::host_vector<double> rCir(NAx);
    ifstream myfile7 ("phantom_rCir.txt", ios::in);
    for (i=0; i<NAx; i++){
        myfile7>>rCir[i];
    }
    myfile7.close();
    
    // The smallest number, which is > NAx, in the base 10
    unsigned int Nmax=0;
    ifstream myfile8 ("phantom_Nmax.txt", ios::in);
    myfile8>>Nmax;
    myfile8.close();
    
    // btable: [bval gx gy gz]
    thrust::host_vector<double> btab(Nbvec*4);
    ifstream myfile9 ("btable.txt", ios::in);
    for (i=0; i<Nbvec*4; i++) {
        myfile9>>btab[i];
    }
    myfile9.close();
    
    // Diffusion time
    thrust::host_vector<double> TD(timepoints);
    for (i=0; i<timepoints; i++){
        TD[i]=(i*(TN/timepoints)+1)*dt;
    }
    
    //********** Initialize Particle Positions in IAS *********
    const double probE=Pi/4.0*stepEX*kappa/Dex;          // Probability constant from ECS to myelin
    const double probI=Pi/4.0*stepIN*kappa/Din;          // Probability constant from ICS to myelin
    stepEX/=res; stepIN/=res;                            // Normalize the step size with the voxel size
    stepEXz/=res; stepINz/=res;
    
    // Create translate flag to speed up the code
    double Lpix = 1.0/static_cast<double>(NPix);
    thrust::host_vector<bool> translateFlag(NAx);
    for (i=0; i<NAx; i++) {
        if ( ((xCir[i]+rCir[i]+2*Lpix)>=1) | ((xCir[i]-rCir[i]-2*Lpix)<=0) | ((yCir[i]+rCir[i]+2*Lpix)>=1) | ((yCir[i]-rCir[i]-2*Lpix)<=0) ) {
            translateFlag[i]=true;
        } else {
            translateFlag[i]=false;
        }
    }
    
    // ********** Simulate diffusion **********
    
    // Initialize seed
    unsigned long seed=0;
    FILE *urandom;
    urandom = fopen("/dev/random", "r");
    fread(&seed, sizeof (seed), 1, urandom);
    fclose(urandom);
    
    // Initialize state of RNG
    int blockSize = thread_per_block;
    int numBlocks = (NPar + blockSize - 1) / blockSize;
    cout<<numBlocks<<endl<<blockSize<<endl;
    
    thrust::device_vector<hiprandStatePhilox4_32_10_t> devState(numBlocks*blockSize);
    setup_kernel<<<numBlocks, blockSize>>>(devState.data().get(),seed);
    
    // Initialize output
    thrust::host_vector<double> dx2(timepoints*6);
    thrust::host_vector<double> dx4(timepoints*15);
    thrust::host_vector<double> NParICS(timepoints);
    thrust::host_vector<double> NParBin(timepoints*nbin);
    thrust::host_vector<double> sig(timepoints*Nbvec);
    for (i=0;i<timepoints*6;i++){ dx2[i]=0; }
    for (i=0;i<timepoints*15;i++){ dx4[i]=0; }
    for (i=0;i<timepoints;i++){ NParICS[i]=0; }
    for (i=0;i<timepoints*nbin;i++){ NParBin[i]=0; }
    for (i=0;i<timepoints*Nbvec;i++) { sig[i]=0; }
    
    // Move data from host to device
    thrust::device_vector<double> d_dx2=dx2;
    thrust::device_vector<double> d_dx4=dx4;
    thrust::device_vector<double> d_NParICS=NParICS;
    thrust::device_vector<double> d_NParBin=NParBin;
    thrust::device_vector<double> d_sig=sig;
    thrust::device_vector<double> d_xCir=xCir;
    thrust::device_vector<double> d_yCir=yCir;
    thrust::device_vector<double> d_rCir=rCir;
    thrust::device_vector<bool> d_translateFlag=translateFlag;
    thrust::device_vector<unsigned int> d_APix=APix;
    thrust::device_vector<double> d_btab=btab;
    thrust::device_vector<double> d_TD=TD;
    
    // Parallel computation
    begin=clock();
    propagate<<<numBlocks, blockSize>>>(devState.data().get(), d_dx2.data().get(), d_dx4.data().get(), d_NParICS.data().get(), d_NParBin.data().get(), d_sig.data().get(), TN, NPar, Nbvec, res, stepIN, stepEX, stepINz, stepEXz, probI, probE, NPix, Nmax, initFlag, d_xCir.data().get(), d_yCir.data().get(), d_rCir.data().get(), d_translateFlag.data().get(), d_APix.data().get(), d_btab.data().get(), d_TD.data().get());
    hipDeviceSynchronize();
    end=clock();
    cout << "Done! Elpased time "<<double((end-begin)/CLOCKS_PER_SEC) << " s"<< endl;
    
    thrust::copy(d_dx2.begin(), d_dx2.end(), dx2.begin());
    thrust::copy(d_dx4.begin(), d_dx4.end(), dx4.begin());
    thrust::copy(d_NParICS.begin(), d_NParICS.end(), NParICS.begin());
    thrust::copy(d_NParBin.begin(), d_NParBin.end(), NParBin.begin());
    thrust::copy(d_sig.begin(), d_sig.end(), sig.begin());
    
    ofstream fdx2out("dx2_diffusion.txt");
    ofstream fdx4out("dx4_diffusion.txt");
    ofstream fNParICSout("NParICS.txt");
    ofstream fNParBinout("NParBin.txt");
    ofstream fsigout("sig_diffusion.txt");
    fdx2out.precision(15);
    fdx4out.precision(15);
    fNParICSout.precision(15);
    fNParBinout.precision(15);
    fsigout.precision(15);
    double dr = 0.5*res/nbin;
    for (i=0; i<timepoints; i++) {
        for (j=0; j<6; j++) {
            if (j==5) {
                fdx2out<<dx2[i*6+j]<<endl;
            } else {
                fdx2out<<dx2[i*6+j]<<"\t";
            }
        }
        for (j=0; j<15; j++) {
            if (j==14) {
                fdx4out<<dx4[i*15+j]<<endl;
            } else {
                fdx4out<<dx4[i*15+j]<<"\t";
            }
        }
        fNParICSout<<NParICS[i]<<endl;
        for (j=0; j<nbin; j++) {
            if (j==nbin-1){
                fNParBinout<<NParBin[i*nbin+j]/(Pi*dr*dr*(2*j+1))<<endl;
            } else {
                fNParBinout<<NParBin[i*nbin+j]/(Pi*dr*dr*(2*j+1))<<"\t";
            }
        }
        for(j=0; j<Nbvec; j++) {
            if (j==Nbvec-1){
                fsigout<<sig[i*Nbvec+j]<<endl;
            } else {
                fsigout<<sig[i*Nbvec+j]<<"\t";
            }
        }
    }
    fdx2out.close();
    fdx4out.close();
    fNParICSout.close();
    fNParBinout.close();
    fsigout.close();
    
    ofstream paraout ("sim_para.txt");
    paraout<<dt<<endl<<TN<<endl<<NPar<<endl<<Nbvec<<endl;
    paraout<<Din<<endl<<Dex<<endl;
    paraout<<kappa<<endl<<initFlag<<endl<<res<<endl;
    paraout.close();
    
    ofstream TDout("diff_time.txt");
    for (i=0; i<timepoints; i++){
        TDout<<(i*(TN/timepoints)+1)*dt<<endl;
    }
    TDout.close();
}

